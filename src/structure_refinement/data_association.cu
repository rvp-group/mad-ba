#include "hip/hip_runtime.h"
#include "data_association.cuh"
#include <stdio.h>
#include <srrg_system_utils/chrono.h>


namespace structure_refinement {

uint64_t Surfelv2::counter_ = 0;

__global__ void associateDataKernel(int numOfLeafs, int kdTreeAIdx, int kdTreeBIdx, TreeNodeTypePtr *kdTreeLeafesPtr, TreeNodeTypePtr *kdTreesPtr, SurfelMatches *matchPtr) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < numOfLeafs; i += stride) {
    // printf("I:  %d ,IndexA: %d, IndexB: %d\n",i, kdTreeAIdx, kdTreeBIdx);
    matchPtr[i].matched = false;
    matchPtr[i].surfelA = kdTreeLeafesPtr[i];
    matchPtr[i].surfelB = kdTreesPtr[kdTreeBIdx]->bestMatchingLeafFast(kdTreeLeafesPtr[i]->mean_);
    TreeNodeTypePtr surfelTmp = kdTreesPtr[kdTreeAIdx]->bestMatchingLeafFast(matchPtr[i].surfelB->mean_);
    // If surfelA <-> is closest to surfelB and vice verse
    if (matchPtr[i].surfelA == surfelTmp) {
      float maxD = 0.3;
      float maxDNorm = 0.9;
      float maxAngle = 5 * M_PI / 180.0;
      float d = (matchPtr[i].surfelA->mean_ - matchPtr[i].surfelB->mean_).norm();
      float dNorm = abs((matchPtr[i].surfelB->mean_ - matchPtr[i].surfelA->mean_).dot(matchPtr[i].surfelA->eigenvectors_.col(0)));
      if (d < maxD || dNorm < maxDNorm) {
        Eigen::Vector3f a = matchPtr[i].surfelA->eigenvectors_.col(0).cast<float>();
        Eigen::Vector3f b = matchPtr[i].surfelB->eigenvectors_.col(0).cast<float>();
        float angle = atan2(a.cross(b).norm(), a.dot(b));
        if (angle < maxAngle)
          matchPtr[i].matched = true;
      }
    }
  }
}

void associateDataKernelCPU(int numOfLeafs, int kdTreeAIdx, int kdTreeBIdx, TreeNodeTypePtr *kdTreeLeafesPtr, TreeNodeTypePtr *kdTreesPtr, SurfelMatches *matchPtr) {
  for (int i = 0; i < numOfLeafs; i += 1) {
    // printf("I:  %d ,IndexA: %d, IndexB: %d\n",i, kdTreeAIdx, kdTreeBIdx);
    matchPtr[i].matched = false;
    matchPtr[i].surfelA = kdTreeLeafesPtr[i];
    matchPtr[i].surfelB = kdTreesPtr[kdTreeBIdx]->bestMatchingLeafFast(kdTreeLeafesPtr[i]->mean_);
    TreeNodeTypePtr surfelTmp = kdTreesPtr[kdTreeAIdx]->bestMatchingLeafFast(matchPtr[i].surfelB->mean_);
    // If surfelA <-> is closest to surfelB and vice verse
    if (matchPtr[i].surfelA == surfelTmp) {
      float maxD = 1 * 1.5;  // 1.5
      float maxDNorm = 1 * 3.0; // 3.0
      float maxAngle = 5 * M_PI / 180.0;  // Smaller value converges faster
      float d = (matchPtr[i].surfelA->mean_ - matchPtr[i].surfelB->mean_).norm();
      float dNorm = abs((matchPtr[i].surfelB->mean_ - matchPtr[i].surfelA->mean_).dot(matchPtr[i].surfelA->eigenvectors_.col(0)));
      if (d < maxD || dNorm < maxDNorm) {
        Eigen::Vector3f a = matchPtr[i].surfelA->eigenvectors_.col(0).cast<float>();
        Eigen::Vector3f b = matchPtr[i].surfelB->eigenvectors_.col(0).cast<float>();
        float angle = atan2(a.cross(b).norm(), a.dot(b));
        if (angle < maxAngle)
          matchPtr[i].matched = true;
      }
    }
  }
}

__host__ void DataAssociation::prepareData(std::vector<std::shared_ptr<TreeNodeType>> &kdTrees, std::vector<std::vector<TreeNodeTypePtr>> &kdTreeLeafes) {
  srrg2_core::Chrono::ChronoMap _timings;
  {
    // Converting shared_ptr to ptr
    std::vector<TreeNodeTypePtr> kdTreePtrs;
    kdTreePtrs.reserve(kdTrees.size());
    TreeNodeTypePtr *devKdTreesPtr;
    srrg2_core::Chrono chGP2wew1("ProcessingMatches", &_timings, false);

    for (int i = 0; i < kdTrees.size(); i++)
      kdTreePtrs.push_back(kdTrees[i].get());

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    // Copy the pointers of kdTrees to GPU (2nd input param)
    size_t dataSize = kdTreePtrs.size() * sizeof(TreeNodeTypePtr);
    {
      srrg2_core::Chrono chGP2("Malloc devKdTreesPtr", &_timings, false);
      hipMalloc(&devKdTreesPtr, dataSize);
      hipMemcpy(devKdTreesPtr, kdTreePtrs.data(), dataSize, hipMemcpyHostToDevice);
    }

    for (int i = 0; i < kdTrees.size() - 1; i++) {

      // Copy the leafs for given kdTree (1st input param)
      TreeNodeTypePtr *devKdTreeLeafesPtr;
      size_t dataSizeLeafs = kdTreeLeafes.at(i).size() * sizeof(TreeNodeTypePtr);
      {
        srrg2_core::Chrono chGP2("Malloc devKdTreeLeafesPtr", &_timings, false);
        hipMalloc(&devKdTreeLeafesPtr, dataSizeLeafs);
        hipMemcpy(devKdTreeLeafesPtr, kdTreeLeafes.at(i).data(), dataSizeLeafs, hipMemcpyHostToDevice);
      }

      // Copy the container for surfelMatches to GPU (output)
      std::vector<SurfelMatches> kdTreeMatches(kdTreeLeafes.at(i).size());
      SurfelMatches *devSurfelMatchesPtr;

      size_t dataSizeMatches = kdTreeMatches.size() * sizeof(SurfelMatches);
      {
        srrg2_core::Chrono chGP2("Malloc devSurfelMatchesPtr", &_timings, false);
        hipMalloc(&devSurfelMatchesPtr, dataSizeMatches);
      }

      // Copy the results back
      // std::vector<SurfelMatches> kdTreeMatches(kdTreeLeafes.at(i).size());
      // // kdTreeMatches.reserve(kdTreeLeafes.at(0).size());
      // SurfelMatches *devSurfelMatchesPtr;

      // size_t dataSizeMatches = kdTreeMatches.size() * sizeof(SurfelMatches);
      // hipMalloc(&devSurfelMatchesPtr, dataSizeMatches);
      for (int j = i + 1; j < kdTrees.size(); j++) {
      // Copy the results back
      // std::vector<SurfelMatches> kdTreeMatches(kdTreeLeafes.at(i).size());
      // // kdTreeMatches.reserve(kdTreeLeafes.at(0).size());
      // SurfelMatches *devSurfelMatchesPtr;

      // size_t dataSizeMatches = kdTreeMatches.size() * sizeof(SurfelMatches);
      // hipMalloc(&devSurfelMatchesPtr, dataSizeMatches);
        {
          srrg2_core::Chrono chGP2("Calling GPU kernel", &_timings, false);
          associateDataKernel<<<32 * numSMs, 256>>>(kdTreeLeafes.at(i).size(), i, j, devKdTreeLeafesPtr, devKdTreesPtr, devSurfelMatchesPtr);
        }
        {
          srrg2_core::Chrono chGP2("Memcopy matches", &_timings, false);
          hipMemcpy(kdTreeMatches.data(), devSurfelMatchesPtr, dataSizeMatches, hipMemcpyDeviceToHost);
        }
        {
          srrg2_core::Chrono chGP2("Processing surfel matches", &_timings, false);
          processTheSurfelMatches(kdTreeMatches);
        }
        std::cout << "KdTree matching " << i << " with " << j << std::endl;
      }
      hipFree(devSurfelMatchesPtr);
      hipFree(devKdTreeLeafesPtr);
    }

  hipDeviceSynchronize();

    // // Free memory
    hipFree(devKdTreesPtr);
  // hipFree(devOutPtr);
  // hipFree(devSurfelMatchesPtr);

  std::cout << "Ended GPU computations " << std::endl;
  // surfels_.clear();
  // Surfelv2::counter_ = 0;
  }
  srrg2_core::Chrono::printReport(_timings);
}

__host__ void DataAssociation::prepareDataCPU(std::vector<std::shared_ptr<TreeNodeType>> & kdTrees, std::vector<std::vector<TreeNodeTypePtr>> & kdTreeLeafes) {


  srrg2_core::Chrono::ChronoMap _timings;
  srrg2_core::Chrono chGP2("ProcessingMatches", &_timings, false);

  std::vector<TreeNodeTypePtr> kdTreePtrs;
  kdTreePtrs.reserve(kdTrees.size());

  for (int i=0; i < kdTrees.size(); i++)
    kdTreePtrs.push_back(kdTrees[i].get());

  for (int i = 0; i < kdTrees.size() - 1; i++) {

    // Copy the container for surfelMatches to GPU (output)
    std::vector<SurfelMatches> kdTreeMatches(kdTreeLeafes.at(i).size());
    for (int j = i + 1; j < kdTrees.size(); j++) {
      std::cout << "KdTree matching " << i << " with " << j << std::endl;
      {
        srrg2_core::Chrono chGP2("Associating data", &_timings, false);
        associateDataKernelCPU(kdTreeLeafes.at(i).size(), i, j, kdTreeLeafes.at(i).data(), kdTreePtrs.data(), kdTreeMatches.data());
      }
      {
        srrg2_core::Chrono chGP2("ProcessingMatches", &_timings, false);
        processTheSurfelMatches(kdTreeMatches);
      }
    }
    int maxNum = 0;
    int idx = 0;
    for (int k = 0; k < surfels_.size(); k++) {
      int num = surfels_.at(k).leafs_.size();
      if (num > maxNum) {
        maxNum = num;
        idx = k;
      }
    }
    std::cout << "Num of surfels " << surfels_.size() << " Max surfels:  " << maxNum << std::endl;
  }
  std::cout << "Ended CPU computations " << std::endl;
          srrg2_core::Chrono::printReport(_timings);

}

// Potentially this can be parallelized, as matches contain only pairs of surfels
__host__ void DataAssociation::processTheSurfelMatches(std::vector<SurfelMatches> &matches) {

  for (int i = 0; i < matches.size(); i++) {
    if (matches[i].matched == false)
      continue;
    int idSurfelA = matches[i].surfelA->surfel_id_;
    int idSurfelB = matches[i].surfelB->surfel_id_;
    if ((idSurfelA != -1) && (idSurfelB != -1)){
    }
    else if (idSurfelA != -1){
      // std::cout << "SurfelA found" << std::endl;
      // Check if given surfel Already has leaf from that pose
      int pointCloudBIdx =  matches[i].surfelB->pointcloud_id_;
      if (surfels_.at(idSurfelA).hasLeafFromPointCloud(pointCloudBIdx) == false)
        surfels_.at(idSurfelA).addLeaf(matches[i].surfelB);
      // std::cout << "Matched1 "  << matches[i].surfelA->pointcloud_id_ << " with "  << matches[i].surfelB->pointcloud_id_ << std::endl;
    }
    else if (idSurfelB != -1){
      // std::cout << "SurfelB found" << std::endl;
      int pointCloudAIdx = matches[i].surfelA->pointcloud_id_;
      if (surfels_.at(idSurfelB).hasLeafFromPointCloud(pointCloudAIdx) == false)
        surfels_.at(idSurfelB).addLeaf(matches[i].surfelA);
      // std::cout << "Matched2 " << matches[i].surfelA->pointcloud_id_ << " with " << matches[i].surfelB->pointcloud_id_ << std::endl;

    }
    else {
      Surfelv2 newSurfel;
      // std::cout << "Surfel ID " << newSurfel.id_ << std::endl;
      newSurfel.addLeaf(matches.at(i).surfelA);
      newSurfel.addLeaf(matches.at(i).surfelB);
      surfels_.push_back(newSurfel);
      // std::cout << "Matched3 " << matches[i].surfelA->pointcloud_id_ << " with " << matches[i].surfelB->pointcloud_id_ << std::endl;
    }
  }
}


__host__ void DataAssociation::prepareDataExample() {
  // int N = 1 << 20;
  // float *x, *y;

  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N * sizeof(float));
  // hipMallocManaged(&y, N * sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // int numSMs;
  // hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  // std::cout << "numSMs: " << numSMs << std::endl;
  // // Run kernel on 1M elements on the GPU
  // associateDataKernel<<<32 * numSMs, 256>>>(N, x, y);
  // // Copy data back
  // // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
  //   maxError = fmax(maxError, fabs(y[i] - 3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);
}



}  // namespace structure_refinement
