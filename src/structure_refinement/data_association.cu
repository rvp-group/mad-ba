#include "hip/hip_runtime.h"
#include "data_association.cuh"
#include <stdio.h>

namespace structure_refinement {

__global__ void associateDataKernel(int n, TreeNodeTypePtr *x) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    Eigen::Vector3d querryPoint(5, 5, 5);
    printf("Index  %d\n", i);
    auto tmp = x[i]->bestMatchingLeafFast(querryPoint);
    printf("Index  %d, point: (%f,%f,%f)\n", i, tmp->mean_[0], tmp->mean_[1], tmp->mean_[2]);
  }
}

__host__ void DataAssociation::prepareData(std::vector<std::shared_ptr<TreeNodeType>> kdTrees) {

  // For test create vector without shared_ptr
  std::vector<TreeNodeTypePtr> kdTreePtrs;
  kdTreePtrs.reserve(kdTrees.size());
  TreeNodeTypePtr * devInPtr;

  for (int i=0; i < kdTrees.size(); i++)
  {
    // kdTreeObjVec[i] = *kdTrees.at(i);
    kdTreePtrs.push_back(kdTrees[i].get());
    printf("Adress: %p\n", kdTreePtrs.data()[i]);
  }

  // Copy the pointers to GPU
  size_t dataSize = kdTrees.size() * sizeof(TreeNodeTypePtr);
  hipMalloc(&devInPtr, dataSize);
  hipMemcpy(devInPtr, kdTreePtrs.data(), dataSize, hipMemcpyHostToDevice);

  Eigen::Vector3d querryPoint(5, 5, 5);
  for (int i = 0; i < kdTreePtrs.size(); i++) {
    auto tmp = kdTreePtrs.at(i)->bestMatchingLeafFast(querryPoint);
    std::cout << "CPU point:  " << tmp->mean_[0] << " " << tmp->mean_[1] << " " << tmp->mean_[2] << std::endl;
  }
  // Call the kernel
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  associateDataKernel<<<1, 256>>>(kdTreePtrs.size(), devInPtr);


  // Alocate space for result
  std::vector<TreeNodeType> kdTreeMatch;
  kdTreeMatch.reserve(kdTrees.size());
  TreeNodeType *devOutPtr;

  // Malloc data on GPU for the kdTree
  // size_t dataSize = kdTrees.size() * sizeof(TreeNodeType);
  // kdTreeObj[0]->doNothing(5);
  // hipMallocManaged((void **)&devInPtr, dataSize);
  // devInPtr->build()

  // hipMallocManaged((void **)&devOutPtr, dataSize);

  // Copy memory
  // hipMemcpy(devInPtr, kdTreeObj.data(), dataSize, hipMemcpyHostToDevice);
  

  // Copy kdTrees leaf by leaf
  // Iterate through all leafs
  // for (int i =0; )
  //hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // hipMallocManaged(&devInPtr, size);
  // hipMallocManaged(&devOutPtr, size);



  // Copy the results back
  // hipMemcpy(kdTreeMatch.data(), devOutPtr ,dataSize, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

    // // Free memory
  hipFree(devInPtr);
  hipFree(devOutPtr);

  std::cout << "Ended GPU computations " << std::endl;
  // Allocate memory on GPU for input
  // hipMalloc(&dx, N *sizeof(float));
  // Allocate memory on GPU for output
  // hipMalloc(&dy, N * sizeof(float));

  // x = (float *)malloc(N * sizeof(float));
  // y = (float *)malloc(N * sizeof(float));

  // For each point in point cloud find the closest one

}

__host__ void DataAssociation::prepareDataExample() {
  // int N = 1 << 20;
  // float *x, *y;

  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N * sizeof(float));
  // hipMallocManaged(&y, N * sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // int numSMs;
  // hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  // std::cout << "numSMs: " << numSMs << std::endl;
  // // Run kernel on 1M elements on the GPU
  // associateDataKernel<<<32 * numSMs, 256>>>(N, x, y);
  // // Copy data back
  // // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
  //   maxError = fmax(maxError, fabs(y[i] - 3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);
}



}  // namespace structure_refinement
