#include "hip/hip_runtime.h"
#include "data_association.cuh"
#include <stdio.h>

namespace structure_refinement {

__global__ void associateDataKernel(int n, TreeNodeType *x, TreeNodeType *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride) {
    Eigen::Vector3d querryPoint(5, 5, 5);
    printf("Index  %d\n", i);
    int b = x[i].doNothing(5);
    printf("Index  %d\n", b);

    x[i].bestTest(querryPoint);
    printf("Index  %d\n", i);

    // if (i == 2)
    // printf("Index  %d, point: (%f,%f,%f)\n", i, tmp->mean_[0], tmp->mean_[1], tmp->mean_[2]);

  }
}

__host__ void DataAssociation::prepareData(std::vector<std::shared_ptr<TreeNodeType>> kdTrees) {

  // For test create vector without shared_ptr
  std::vector<TreeNodeType> kdTreeObj;
  kdTreeObj.reserve(kdTrees.size());
  TreeNodeType *devInPtr;

  for (int i=0; i < kdTrees.size(); i++)
    kdTreeObj.push_back(*kdTrees.at(i));

  // Alocate space for result
  std::vector<TreeNodeType> kdTreeMatch;
  kdTreeMatch.reserve(kdTrees.size());
  TreeNodeType *devOutPtr;

  // Malloc data on GPU
  size_t dataSize = kdTrees.size() * sizeof(TreeNodeType);
  hipMalloc((void **)&devInPtr, dataSize); 
  hipMalloc((void **)&devOutPtr, dataSize);

  // Copy memory
  hipMemcpy(devInPtr, kdTreeObj.data(), dataSize, hipMemcpyHostToDevice);
  //hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // hipMallocManaged(&devInPtr, size);
  // hipMallocManaged(&devOutPtr, size);

  // Call the kernel
   int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  associateDataKernel<<<1, 256>>>(kdTreeObj.size(), devInPtr, devOutPtr);

  // Copy the results back
  hipMemcpy(kdTreeMatch.data(), devOutPtr ,dataSize, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

    // // Free memory
  hipFree(devInPtr);
  hipFree(devOutPtr);

  std::cout << "Ended GPU computations " << std::endl;
  // Allocate memory on GPU for input
  // hipMalloc(&dx, N *sizeof(float));
  // Allocate memory on GPU for output
  // hipMalloc(&dy, N * sizeof(float));

  // x = (float *)malloc(N * sizeof(float));
  // y = (float *)malloc(N * sizeof(float));

  // For each point in point cloud find the closest one

}

__host__ void DataAssociation::prepareDataExample() {
  // int N = 1 << 20;
  // float *x, *y;

  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N * sizeof(float));
  // hipMallocManaged(&y, N * sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // int numSMs;
  // hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  // std::cout << "numSMs: " << numSMs << std::endl;
  // // Run kernel on 1M elements on the GPU
  // associateDataKernel<<<32 * numSMs, 256>>>(N, x, y);
  // // Copy data back
  // // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
  //   maxError = fmax(maxError, fabs(y[i] - 3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);
}



}  // namespace structure_refinement
